
#include <hip/hip_runtime.h>
#ifdef CUDA

#include "coulomb_kernel.h"

static void HandleError( cudaError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != cudaSuccess)
    {
		printf( "%s in %s at line %d\n", cudaGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

namespace Kernel {

    /**
     * @brief Perpendicular projection operator Q (eq. 10)
     * 
     * @param ret 
     * @param v 
     */
    __device__ void Q(Matrix2d* ret, Vector2d v) {
        double norm2 = v.squaredNorm();
        if (norm2 < 1E-14) {
            ret->setZero();
        } else {
            ret->coeffRef(0,0) = - v(0)*v(0) / norm2 + 1;
            ret->coeffRef(1,1) = - v(1)*v(1) / norm2 + 1;
            ret->coeffRef(1,0) = - v(1)*v(0) / norm2;
            ret->coeffRef(0,1) = ret->coeffRef(1,0);
            *ret /= sqrt(norm2);
        }
    }

    /**
     * @brief Build the product of Q(v_p^{n+1/2} - v_p'^{n+1/2}) and /Gamma(S_eps^n, p, p')
     * 
     * @param ret 
     * @param p0 
     * @param p1 
     * @param config 
     */
    __global__ void cuda_f_eq_motion(
        double* ret,
        Particle2d* p0,
        Particle2d* p1,
        double* dSdV,
        Config* config
    ) {
        Vector2d gammaTmp;
        Matrix2d qTmp;

        int i = blockIdx.x*config->cudaThreadsPerBlock + threadIdx.x;
        if (i < config->nmarkers) {
            ret[2*i] = 0;
            ret[2*i+1] = 0;
            for (int j=0; j<config->nmarkers; j++) {
                if (i!=j) {
                    Q(&qTmp, (p1[i].z + p0[i].z - p1[j].z - p0[j].z) / 2);
                    gammaTmp(0) = dSdV[2*i] - dSdV[2*j];
                    gammaTmp(1) = dSdV[2*i+1] - dSdV[2*j+1];
                    gammaTmp = qTmp*gammaTmp;
                    ret[2*i] += config->nu / config->m * p1[j].weight * gammaTmp(0);
                    ret[2*i+1] += config->nu / config->m * p1[j].weight * gammaTmp(1);
                }
            }
        }
    }

    __global__ void cuda_dSdv(
        double* ret,
        Particle2d* p,
        Config* config 
    ) {
        int i_p1 = blockIdx.x*config->cudaThreadsPerBlock + threadIdx.x;
        if (i_p1 < config->nmarkers) {
            double logsum, k;
            for (int i_x = 0; i_x < 2; i_x++) {
                ret[2*i_p1+i_x] = 0;
                for (int i=0; i<config->nHermite; i++)
                for (int j=0; j<config->nHermite; j++) {
                    logsum = 0;
                    for (int i_p2 = 0; i_p2<config->nmarkers; i_p2++) {
                        logsum += p[i_p2].weight / (CONST_2PI*config->eps)* ( exp(
                                    -pow(config->kHermite[i] + (p[i_p1].z[0] - p[i_p2].z[0])/ sqrt(2*config->eps), 2)
                                    -pow(config->kHermite[j] + (p[i_p1].z[1] - p[i_p2].z[1])/ sqrt(2*config->eps), 2))
                                );
                    }
                    logsum = log(logsum);
                    if (i_x == 0) {
                        k = config->kHermite[i];
                    } else {
                        k = config->kHermite[j];
                    }
                    ret[2*i_p1+i_x] += k * config->wHermite[i] * config->wHermite[j] * (1. + logsum);
                }
                ret[2*i_p1+i_x] *= sqrt(2.*config->eps) / (config->m * CONST_PI * config->eps);
            }
        }
    }

    /**
     * @brief Compute dv of the equations of motion
     * 
     * @param dv 
     * @param p0 
     * @param p1 
     * @param dSdV 
     * @param config 
     */
    void f_eqmotion_dv(
        VectorXd* dv,
        Particle2d* p0,
        Particle2d* p1,
        VectorXd* dSdV,
        Config* config
    ) {
        // CUDA blocks configuration
        int nblocks = ceil(float(config->nmarkers) / config->cudaThreadsPerBlock);

        // // Allocate device arrays
        double *d_ret, *d_dSdv;
        Config* d_config;
        Particle2d *d_p0, *d_p1;
        HANDLE_ERROR(cudaMalloc((void **)&d_ret, sizeof(double)*2*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_dSdv, sizeof(double)*2*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_config, sizeof(Config)));
        HANDLE_ERROR(cudaMalloc((void **)&d_p0, sizeof(Particle2d)*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_p1, sizeof(Particle2d)*config->nmarkers));

        // Copy to device
        HANDLE_ERROR(cudaMemcpy(d_config, config, sizeof(Config), cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_dSdv, dSdV->data(), sizeof(double)*dSdV->size(), cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_p0, p0, sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_p1, p1, sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));

        // Compute the entropy gradient
        // cudaThreadSynchronize();
        cuda_f_eq_motion<<<nblocks, config->cudaThreadsPerBlock>>>(d_ret, d_p0, d_p1, d_dSdv, d_config);
        HANDLE_ERROR( cudaPeekAtLastError() );
        // cudaThreadSynchronize();
        
        // Copy to host
        HANDLE_ERROR(cudaMemcpy(dv->data(), d_ret, sizeof(double)*dv->size(), cudaMemcpyDeviceToHost));
    }


    void computedSdv(
        VectorXd* ret,
        Particle2d* p,
        Config* config
    ) {

        // CUDA blocks configuration
        int nblocks = ceil(float(config->nmarkers) / config->cudaThreadsPerBlock);

        // // Allocate device arrays
        double *d_ret;
        Config* d_config, *l_config = new Config;
        Particle2d* d_p;
        double* d_kHermite, *d_wHermite;
        Vector2d *d_u1, *d_u2;
        HANDLE_ERROR(cudaMalloc((void **)&d_ret, sizeof(double)*2*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_config, sizeof(Config)));
        HANDLE_ERROR(cudaMalloc((void **)&d_p, sizeof(Particle2d)*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_kHermite, sizeof(double)*config->nHermite));
        HANDLE_ERROR(cudaMalloc((void **)&d_wHermite, sizeof(double)*config->nHermite));
        HANDLE_ERROR(cudaMalloc((void **)&d_u1, sizeof(Vector2d)));
        HANDLE_ERROR(cudaMalloc((void **)&d_u2, sizeof(Vector2d)));

        // // Copy to device
        memcpy(l_config, config, sizeof(Config));
        HANDLE_ERROR(cudaMemcpy(d_kHermite, config->kHermite, sizeof(double)*config->nHermite, cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_wHermite, config->wHermite, sizeof(double)*config->nHermite, cudaMemcpyHostToDevice));
        l_config->kHermite = d_kHermite;
        l_config->wHermite = d_wHermite;
        HANDLE_ERROR(cudaMemcpy(d_config, l_config, sizeof(Config), cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_p, p, sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));

        // // Compute the entropy gradient
        // cudaThreadSynchronize();
        // printf("computing entropy\n");
        cuda_dSdv<<<nblocks, config->cudaThreadsPerBlock>>>(d_ret, d_p, d_config);
        HANDLE_ERROR( cudaPeekAtLastError() );
        // cudaThreadSynchronize();
        
        // Copy to host
        HANDLE_ERROR(cudaMemcpy(ret->data(), d_ret, sizeof(double)*ret->size(), cudaMemcpyDeviceToHost));
    }
}

#endif