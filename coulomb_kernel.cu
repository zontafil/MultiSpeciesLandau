
#include <hip/hip_runtime.h>
#ifdef CUDA

#include "coulomb_kernel.h"

static void HandleError( cudaError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != cudaSuccess)
    {
		printf( "%s in %s at line %d\n", cudaGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

namespace Kernel {

    /**
     * @brief Perpendicular projection operator Q (eq. 10)
     * 
     * @param ret 
     * @param v 
     */
    __device__ void Q(Matrix2d* ret, Vector2d v) {
        double norm2 = v.squaredNorm();
        if (norm2 < 1E-14) {
            ret->setZero();
        } else {
            ret->coeffRef(0,0) = - v(0)*v(0) / norm2 + 1;
            ret->coeffRef(1,1) = - v(1)*v(1) / norm2 + 1;
            ret->coeffRef(1,0) = - v(1)*v(0) / norm2;
            ret->coeffRef(0,1) = ret->coeffRef(1,0);
            *ret /= sqrt(norm2);
        }
    }

    /**
     * @brief Build the product of Q(v_p^{n+1/2} - v_p'^{n+1/2}) and /Gamma(S_eps^n, p, p')
     * 
     * @param ret 
     * @param p0 
     * @param p1 
     * @param config 
     */
    __global__ void cuda_f_eq_motion(
        double** ret,
        Particle2d** p0,
        Particle2d** p1,
        double** dSdV,
        Config* config,
        Specie* species
    ) {
        Vector2d gammaTmp, qGamma, z0_is, z1_is, dSdV_is;
        Matrix2d qTmp;
        Particle2d *p0_s2, *p1_s2;
        double nu, m;
        double *dSdV_s2;
        double *ret_s;

        // this thread is computing the EOM contribution of marker i in specie s
        // we perform a sum in all the markers j of all the species s2
        int global_idx = blockIdx.x*config->cudaThreadsPerBlock + threadIdx.x;
        int s = global_idx / config->nmarkers;
        int i = global_idx % config->nmarkers;
        if (s < config->nspecies) {

            // set some pointer to s block, to avoid doing additional computations
            ret[s][2*i] = 0;
            ret[s][2*i+1] = 0;
            ret_s = ret[s];
            z0_is = p0[s][i].z;
            z1_is = p1[s][i].z;
            dSdV_is(0) = dSdV[s][2*i];
            dSdV_is(1) = dSdV[s][2*i+1];
            m = species[s].m;
            for (int s2=0; s2<config->nspecies; s2++) {
                // set some pointer to s2 block, to avoid doing additional computations
                p0_s2 = p0[s2];
                p1_s2 = p1[s2];
                nu = species[s].nu[s2];
                dSdV_s2 = dSdV[s2];
                for (int j=0; j<config->nmarkers; j++) {
                    if (i!=j || s!=s2) {
                        Q(&qTmp, (z1_is + z0_is - p1_s2[j].z - p0_s2[j].z) / 2);
                        gammaTmp(0) = dSdV_is(0) - dSdV_s2[2*j];
                        gammaTmp(1) = dSdV_is(1) - dSdV_s2[2*j+1];
                        gammaTmp = qTmp*gammaTmp;
                        ret_s[2*i] -= nu / m * p1_s2[j].weight * gammaTmp(0);
                        ret_s[2*i+1] -= nu / m * p1_s2[j].weight * gammaTmp(1);
                    }
                }
            }
        }
    }

    /**
     * @brief CUDA version of entropy gradient
     */
    __global__ void cuda_dSdv(
        double** ret,
        Particle2d** p,
        Config* config,
        Specie* species
    ) {
        int global_idx = blockIdx.x*config->cudaThreadsPerBlock + threadIdx.x;
        int s = global_idx / config->nmarkers;
        int s2 = s;
        int i_p1 = global_idx % config->nmarkers;
        if (s < config->nspecies) {
            Particle2d* ps1 = p[s];
            double* rets1 = ret[s];
            double m = species[s].m;
            double logsum, dx, dy, kpx, kpy;
            double SQRT2EPSM1 = 1./sqrt(2.*config->eps);
            double PI2EPSM1 = 1./(CONST_2PI * config->eps);
            rets1[2*i_p1] = 0;
            rets1[2*i_p1+1] = 0;
            for (int i=0; i<config->nHermite; i++)
            for (int j=0; j<config->nHermite; j++) {
                logsum = 0;

                // TODO: Normalize z to SQRT2EPSM1 --> ~10% performance boost
                kpx = config->kHermite[i] + ps1[i_p1].z[0] * SQRT2EPSM1;
                kpy = config->kHermite[j] + ps1[i_p1].z[1] * SQRT2EPSM1;

                for (int i_p2 = 0; i_p2<config->nmarkers; i_p2++) {
                    dx = kpx - p[s2][i_p2].z[0] * SQRT2EPSM1;
                    dy = kpy - p[s2][i_p2].z[1] * SQRT2EPSM1;
                    logsum+=p[s2][i_p2].weight* exp(-dx*dx - dy*dy);
                }
                logsum = config->wHermite[i]*config->wHermite[j] * (1. + log(logsum * PI2EPSM1));
                rets1[2*i_p1] += logsum * config->kHermite[i];
                rets1[2*i_p1+1] += logsum * config->kHermite[j];
            }
            rets1[2*i_p1] *= sqrt(2.*config->eps) / (m * CONST_PI * config->eps);
            rets1[2*i_p1+1] *= sqrt(2.*config->eps) / (m * CONST_PI * config->eps);
        }
    }

    /**
     * @brief Compute dv of the equations of motion
     * 
     * @param dv 
     * @param p0 
     * @param p1 
     * @param dSdV 
     * @param config 
     */
    void f_eqmotion_dv(
        VectorXd* dv,
        Particle2d** p0,
        Particle2d** p1,
        VectorXd* dSdV,
        Config* config
    ) {
        // CUDA blocks configuration
        int nblocks = ceil(float(config->nmarkers * config->nspecies) / config->cudaThreadsPerBlock);

        // init particles
        Particle2d** h_p0 = new Particle2d*[config->nspecies];
        Particle2d** h_p1 = new Particle2d*[config->nspecies];
        Particle2d** d_p0, **d_p1;
        HANDLE_ERROR(cudaMalloc((void **)&d_p0, sizeof(Particle2d*)*config->nspecies));    
        HANDLE_ERROR(cudaMalloc((void **)&d_p1, sizeof(Particle2d*)*config->nspecies));    
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMalloc((void **)&(h_p0[s]), sizeof(Particle2d)*config->nmarkers));
            HANDLE_ERROR(cudaMalloc((void **)&(h_p1[s]), sizeof(Particle2d)*config->nmarkers));
            HANDLE_ERROR(cudaMemcpy(h_p0[s], p0[s], sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));
            HANDLE_ERROR(cudaMemcpy(h_p1[s], p1[s], sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));
        }
        HANDLE_ERROR(cudaMemcpy (d_p0, h_p0, config->nspecies*sizeof(Particle2d*), cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy (d_p1, h_p1, config->nspecies*sizeof(Particle2d*), cudaMemcpyHostToDevice));

        // init config
        Config* d_config;
        HANDLE_ERROR(cudaMalloc((void **)&d_config, sizeof(Config)));
        HANDLE_ERROR(cudaMemcpy(d_config, config, sizeof(Config), cudaMemcpyHostToDevice));

        // init spieces config
        Specie h_species[config->nspecies], *d_species;
        HANDLE_ERROR(cudaMalloc((void **)&d_species, sizeof(Specie)*config->nspecies));    
        for (int s=0; s<config->nspecies; s++) {
            memcpy(&h_species[s], &config->species[s], sizeof(Specie));
            HANDLE_ERROR(cudaMalloc((void **)&(h_species[s].nu), sizeof(double)*config->nspecies));    
            HANDLE_ERROR(cudaMemcpy(h_species[s].nu, config->species[s].nu, sizeof(double)*config->nspecies, cudaMemcpyHostToDevice));
        }
        HANDLE_ERROR(cudaMemcpy (d_species, h_species, config->nspecies*sizeof(Specie), cudaMemcpyHostToDevice));

        // init entropy
        double** h_dSdv = new double*[config->nspecies], **d_dSdv;
        HANDLE_ERROR(cudaMalloc((void **)&d_dSdv, sizeof(double*)*config->nspecies));    
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMalloc((void **)&h_dSdv[s], sizeof(double)*2*config->nmarkers));
            HANDLE_ERROR(cudaMemcpy(h_dSdv[s], dSdV[s].data(), sizeof(double)*dSdV[s].size(), cudaMemcpyHostToDevice));
        }
        HANDLE_ERROR(cudaMemcpy (d_dSdv, h_dSdv, config->nspecies*sizeof(double*), cudaMemcpyHostToDevice));

        // init ret
        double** h_ret = new double*[config->nspecies], **d_ret;
        HANDLE_ERROR(cudaMalloc((void **)&d_ret, sizeof(double*)*config->nspecies));    
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMalloc((void **)&h_ret[s], sizeof(double)*2*config->nmarkers));
        }
        HANDLE_ERROR(cudaMemcpy (d_ret, h_ret, config->nspecies*sizeof(double*), cudaMemcpyHostToDevice));

        // Compute the entropy gradient
        cuda_f_eq_motion<<<nblocks, config->cudaThreadsPerBlock>>>(d_ret, d_p0, d_p1, d_dSdv, d_config, d_species);
        HANDLE_ERROR( cudaPeekAtLastError() );
        
        // Copy to host
        HANDLE_ERROR(cudaMemcpy(h_ret, d_ret, sizeof(double)*config->nspecies, cudaMemcpyDeviceToHost));
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMemcpy(dv[s].data(), h_ret[s], sizeof(double)*dv[s].size(), cudaMemcpyDeviceToHost));
        }
    }


    void computedSdv(
        VectorXd* ret,
        Particle2d** p,
        Config* config
    ) {

        // CUDA blocks configuration
        int nblocks = ceil(float(config->nmarkers) / config->cudaThreadsPerBlock);

        // // Allocate device arrays
        Config* d_config, *l_config = new Config;
        Particle2d** d_p;
        Particle2d** h_p = new Particle2d*[config->nspecies];

        double* d_kHermite, *d_wHermite;
        HANDLE_ERROR(cudaMalloc((void **)&d_p, sizeof(Particle2d*)*config->nspecies));    
        HANDLE_ERROR(cudaMalloc((void **)&d_config, sizeof(Config)));
        HANDLE_ERROR(cudaMalloc((void **)&d_kHermite, sizeof(double)*config->nHermite));
        HANDLE_ERROR(cudaMalloc((void **)&d_wHermite, sizeof(double)*config->nHermite));
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMalloc((void **)&(h_p[s]), sizeof(Particle2d)*config->nmarkers));
            HANDLE_ERROR(cudaMemcpy(h_p[s], p[s], sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));
        }
        HANDLE_ERROR(cudaMemcpy (d_p, h_p, config->nspecies*sizeof(Particle2d*), cudaMemcpyHostToDevice));

        // init ret
        double** h_ret = new double*[config->nspecies], **d_ret;
        HANDLE_ERROR(cudaMalloc((void **)&d_ret, sizeof(double*)*config->nspecies));    
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMalloc((void **)&h_ret[s], sizeof(double)*2*config->nmarkers));
        }
        HANDLE_ERROR(cudaMemcpy (d_ret, h_ret, config->nspecies*sizeof(double*), cudaMemcpyHostToDevice));

        // init spieces config
        Specie h_species[config->nspecies], *d_species;
        HANDLE_ERROR(cudaMalloc((void **)&d_species, sizeof(Specie)*config->nspecies));    
        for (int s=0; s<config->nspecies; s++) {
            memcpy(&h_species[s], &config->species[s], sizeof(Specie));
            HANDLE_ERROR(cudaMalloc((void **)&(h_species[s].nu), sizeof(double)*config->nspecies));    
            HANDLE_ERROR(cudaMemcpy(h_species[s].nu, config->species[s].nu, sizeof(double)*config->nspecies, cudaMemcpyHostToDevice));
        }
        HANDLE_ERROR(cudaMemcpy (d_species, h_species, config->nspecies*sizeof(Specie), cudaMemcpyHostToDevice));

        // // Copy to device
        memcpy(l_config, config, sizeof(Config));
        HANDLE_ERROR(cudaMemcpy(d_kHermite, config->kHermite, sizeof(double)*config->nHermite, cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_wHermite, config->wHermite, sizeof(double)*config->nHermite, cudaMemcpyHostToDevice));
        l_config->kHermite = d_kHermite;
        l_config->wHermite = d_wHermite;
        HANDLE_ERROR(cudaMemcpy(d_config, l_config, sizeof(Config), cudaMemcpyHostToDevice));

        // for (int s=0; s<config->nspecies; s++) {
        // // Compute the entropy gradient
        // HANDLE_ERROR(cudaMemcpy(d_p, p[s], sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));
        // HANDLE_ERROR(cudaMemcpy(d_p2, p[1], sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));
        cuda_dSdv<<<nblocks, config->cudaThreadsPerBlock>>>(d_ret, d_p, d_config, d_species);
        HANDLE_ERROR( cudaPeekAtLastError() );

        // Copy to host
        for (int s=0; s<config->nspecies; s++) {
            HANDLE_ERROR(cudaMemcpy(ret[s].data(), h_ret[s], sizeof(double)*ret[s].size(), cudaMemcpyDeviceToHost));
        }
        // HANDLE_ERROR(cudaMemcpy(ret[s].data(), d_ret, sizeof(double)*ret[s].size(), cudaMemcpyDeviceToHost));
        // }
    }
}

#endif