
#include <hip/hip_runtime.h>
#ifdef CUDA

#include "coulomb_kernel.h"

static void HandleError( cudaError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != cudaSuccess)
    {
		printf( "%s in %s at line %d\n", cudaGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

namespace Kernel {

    __global__ void cuda_dSdv(
        double* ret,
        Particle2d* p,
        Config* config 
    ) {
        int i_p1 = blockIdx.x*config->cudaThreadsPerBlock + threadIdx.x;
        if (i_p1 < config->nmarkers) {
            double logsum, k;
            for (int i_x = 0; i_x < 2; i_x++) {
                ret[2*i_p1+i_x] = 0;
                for (int i=0; i<config->nHermite; i++)
                for (int j=0; j<config->nHermite; j++) {
                    logsum = 0;
                    for (int i_p2 = 0; i_p2<config->nmarkers; i_p2++) {
                        logsum += p[i_p2].weight / (CONST_2PI*config->eps)* ( exp(
                                    -pow(config->kHermite[i] + (p[i_p1].z[0] - p[i_p2].z[0])/ sqrt(2*config->eps), 2)
                                    -pow(config->kHermite[j] + (p[i_p1].z[1] - p[i_p2].z[1])/ sqrt(2*config->eps), 2))
                                );
                    }
                    logsum = log(logsum);
                    if (i_x == 0) {
                        k = config->kHermite[i];
                    } else {
                        k = config->kHermite[j];
                    }
                    ret[2*i_p1+i_x] += k * config->wHermite[i] * config->wHermite[j] * (1. + logsum);
                }
                ret[2*i_p1+i_x] *= sqrt(2.*config->eps) / (config->m * CONST_PI * config->eps);
            }
        }
    }


    void computedSdv(
        VectorXd* ret,
        Particle2d* p,
        Config* config
    ) {

        // CUDA blocks configuration
        int nblocks = ceil(float(config->nmarkers) / config->cudaThreadsPerBlock);

        // // Allocate device arrays
        double *d_ret;
        Config* d_config, *l_config = new Config;
        Particle2d* d_p;
        double* d_kHermite, *d_wHermite;
        Vector2d *d_u1, *d_u2;
        HANDLE_ERROR(cudaMalloc((void **)&d_ret, sizeof(double)*2*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_config, sizeof(Config)));
        HANDLE_ERROR(cudaMalloc((void **)&d_p, sizeof(Particle2d)*config->nmarkers));
        HANDLE_ERROR(cudaMalloc((void **)&d_kHermite, sizeof(double)*config->nHermite));
        HANDLE_ERROR(cudaMalloc((void **)&d_wHermite, sizeof(double)*config->nHermite));
        HANDLE_ERROR(cudaMalloc((void **)&d_u1, sizeof(Vector2d)));
        HANDLE_ERROR(cudaMalloc((void **)&d_u2, sizeof(Vector2d)));

        // // Copy to device
        memcpy(l_config, config, sizeof(Config));
        HANDLE_ERROR(cudaMemcpy(d_kHermite, config->kHermite, sizeof(double)*config->nHermite, cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_wHermite, config->wHermite, sizeof(double)*config->nHermite, cudaMemcpyHostToDevice));
        l_config->kHermite = d_kHermite;
        l_config->wHermite = d_wHermite;
        HANDLE_ERROR(cudaMemcpy(d_config, l_config, sizeof(Config), cudaMemcpyHostToDevice));
        HANDLE_ERROR(cudaMemcpy(d_p, p, sizeof(Particle2d)*config->nmarkers, cudaMemcpyHostToDevice));

        // // Compute the entropy gradient
        cudaThreadSynchronize();
        printf("computing entropy\n");
        cuda_dSdv<<<nblocks, config->cudaThreadsPerBlock>>>(d_ret, d_p, d_config);
        HANDLE_ERROR( cudaPeekAtLastError() );
        cudaThreadSynchronize();
        
        // Copy to host
        HANDLE_ERROR(cudaMemcpy(ret->data(), d_ret, sizeof(double)*ret->size(), cudaMemcpyDeviceToHost));
    }
}

#endif